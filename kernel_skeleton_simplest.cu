#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

#define SAMPLE_MAX_LEN 200000
#define SIGNATURE_MAX_LEN 10000

__global__ void getMatches(const char* __restrict d_sample_seq, const char* __restrict d_sample_qual, const char* __restrict d_signature_seq,
        double* __restrict match_scores) {

    const char *sample = &d_sample_seq[SAMPLE_MAX_LEN * blockIdx.x];
    const char *signature = &d_signature_seq[SIGNATURE_MAX_LEN * threadIdx.x];

    for (int i = 0; i < SAMPLE_MAX_LEN; ++i) {
        if (!sample[i]) { // Sample ended; no match
            return;
        }

        for (int j = 0; j < SIGNATURE_MAX_LEN + 1; ++j) { // +1 iteration for when signature len == SIGNATURE_MAX_LEN
            if (j == SIGNATURE_MAX_LEN || !signature[j]) { // Signature ended; found match
                const char *qual = &d_sample_qual[SAMPLE_MAX_LEN * blockIdx.x + i];
                int tot = 0;
                for (int s = 0; s < j; ++s) {
                    tot += qual[s] - 33;
                }

                match_scores[blockIdx.x * blockDim.x + threadIdx.x] = (double)tot / j;
                return;

            } else if (!sample[i + j] || sample[i + j] != 'N' && signature[j] != 'N' &&
                    sample[i + j] != signature[j]) { // No match
                break;
            }
        }
    }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    int sample_arr_len = samples.size() * SAMPLE_MAX_LEN;
    int signature_arr_len = signatures.size() * SIGNATURE_MAX_LEN;

    // Device variables
    char *d_sample_seq, *d_sample_qual, *d_signature_seq;
    int *match_count; // 1 int storing the number of matches
    hipMallocAsync(&d_sample_seq, sample_arr_len, stream1);
    hipMemsetAsync(d_sample_seq, 0, sample_arr_len, stream1); // Initialise arrays to '\0'

    hipMallocAsync(&d_sample_qual, sample_arr_len, stream2);
    hipMemsetAsync(d_sample_qual, 0, sample_arr_len, stream2);

    hipMallocAsync(&d_signature_seq, signature_arr_len, stream3);
    hipMemsetAsync(d_signature_seq, 0, signature_arr_len, stream3);

    hipMallocAsync(&match_count, sizeof(int), stream3);
    hipMemsetAsync(match_count, 0, 1, stream3); // Initialise match_count to 0

    // Pinned memory
    double *match_scores;      // [Score_1, Score2, ...]

    for (int i = 0; i < signatures.size(); ++i) {
        hipMemcpyAsync(&d_signature_seq[i * SIGNATURE_MAX_LEN], signatures[i].seq.c_str(), signatures[i].seq.size(), hipMemcpyHostToDevice, stream3);
    }
    for (int i = 0; i < samples.size(); ++i) {
        hipMemcpyAsync(&d_sample_seq[i * SAMPLE_MAX_LEN], samples[i].seq.c_str(), samples[i].seq.size(), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(&d_sample_qual[i * SAMPLE_MAX_LEN], samples[i].qual.c_str(), samples[i].qual.size(), hipMemcpyHostToDevice, stream2);
    }
    

    // Use hipHostMalloc() to write straight to host, since only a small number of matches should be found
    hipHostMalloc(&match_scores, sizeof(double) * samples.size() * signatures.size()); // Max possible number of matches
    memset(match_scores, 0, sizeof(double) * samples.size() * signatures.size());

    // 1 Sample per block; Each thread in block corresponds to 1 signature
    getMatches<<<samples.size(), signatures.size()>>>(d_sample_seq, d_sample_qual, d_signature_seq, match_scores);

    hipFreeAsync(d_sample_seq, stream1);
    hipFreeAsync(d_sample_qual, stream2);
    hipFreeAsync(d_signature_seq, stream3);
    hipDeviceSynchronize();
    for (int i = 0; i < samples.size() * signatures.size(); ++i) {
        if (match_scores[i] != 0) {
            matches.emplace_back(MatchResult(samples[i / signatures.size()].name,
                                             signatures[i % signatures.size()].name,
                                             match_scores[i]));
        }
    }
    
    hipHostFree(match_scores);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
}
