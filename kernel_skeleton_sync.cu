#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

#define SAMPLE_MAX_LEN 200000
#define SIGNATURE_MAX_LEN 10000

__global__ void getMatches(const char* __restrict d_sample_seq, const char* __restrict d_sample_qual, const char* __restrict d_signature_seq,
        double* __restrict match_scores, unsigned short* __restrict match_idx, int* __restrict match_count) {

    const char *sample = &d_sample_seq[SAMPLE_MAX_LEN * blockIdx.x];
    const char *signature = &d_signature_seq[SIGNATURE_MAX_LEN * threadIdx.x];

    for (int i = 0; i < SAMPLE_MAX_LEN; ++i) {
        if (!sample[i]) { // Sample ended; no match
            return;
        }

        for (int j = 0; j < SIGNATURE_MAX_LEN + 1; ++j) { // +1 iteration for when signature len == SIGNATURE_MAX_LEN
            if (j == SIGNATURE_MAX_LEN || !signature[j]) { // Signature ended; found match
                const char *qual = &d_sample_qual[SAMPLE_MAX_LEN * blockIdx.x + i];
                int tot = 0;
                for (int s = 0; s < j; ++s) {
                    tot += qual[s] - 33;
                }

                int idx = atomicAdd(match_count, 1); // Each idx will be unique (no need synchronisation)
                match_scores[idx] = (double)tot / j;
                match_idx[idx << 1] = blockIdx.x; // Sample idx
                match_idx[(idx << 1) + 1] = threadIdx.x; // Signature idx
                return;

            } else if (!sample[i + j] || sample[i + j] != 'N' && signature[j] != 'N' &&
                    sample[i + j] != signature[j]) { // No match
                break;
            }
        }
    }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    int sample_arr_len = samples.size() * SAMPLE_MAX_LEN;
    int signature_arr_len = signatures.size() * SIGNATURE_MAX_LEN;

    // Device variables
    char *d_sample_seq, *d_sample_qual, *d_signature_seq;
    int *match_count; // 1 int storing the number of matches
    hipMalloc(&d_sample_seq, sample_arr_len);
    hipMemset(d_sample_seq, 0, sample_arr_len); // Initialise arrays to '\0'

    hipMalloc(&d_sample_qual, sample_arr_len);
    hipMemset(d_sample_qual, 0, sample_arr_len);

    hipMalloc(&d_signature_seq, signature_arr_len);
    hipMemset(d_signature_seq, 0, signature_arr_len);

    hipMalloc(&match_count, sizeof(int));
    hipMemset(match_count, 0, 1); // Initialise match_count to 0

    // Pinned memory
    double *match_scores;      // [Score_1, Score2, ...]
    unsigned short *match_idx; // [Samp_idx_1, Sig_idx_1, Samp_idx_2, Sig_idx_2, ...]

    for (int i = 0; i < samples.size(); ++i) {
        hipMemcpy(&d_sample_seq[i * SAMPLE_MAX_LEN], samples[i].seq.c_str(), samples[i].seq.size(), hipMemcpyHostToDevice);
        hipMemcpy(&d_sample_qual[i * SAMPLE_MAX_LEN], samples[i].qual.c_str(), samples[i].qual.size(), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < signatures.size(); ++i) {
        hipMemcpy(&d_signature_seq[i * SIGNATURE_MAX_LEN], signatures[i].seq.c_str(), signatures[i].seq.size(), hipMemcpyHostToDevice);
    }

    // Use hipHostMalloc() to write straight to host, since only a small number of matches should be found
    hipHostMalloc(&match_scores, sizeof(double) * samples.size() * signatures.size()); // Max possible number of matches
    hipHostMalloc(&match_idx, sizeof(unsigned short) * 2 * samples.size() * signatures.size());
    // Test out zero-copy memory
    // hipHostAlloc(&match_scores, sizeof(double) * samples.size() * signatures.size(), hipHostMallocMapped); // Max possible number of matches
    // hipHostAlloc(&match_idx, sizeof(unsigned short) * 2 * samples.size() * signatures.size(), hipHostMallocMapped);
    

    // 1 Sample per block; Each thread in block corresponds to 1 signature
    getMatches<<<samples.size(), signatures.size()>>>(d_sample_seq, d_sample_qual, d_signature_seq, match_scores, match_idx, match_count);

    int h_match_count; // Number of matches after getMatches() is done
    hipMemcpy(&h_match_count, match_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(match_count);
    hipFree(d_sample_seq);
    hipFree(d_sample_qual);
    hipFree(d_signature_seq);

    for (int i = 0; i < h_match_count; ++i) {
        matches.emplace_back(MatchResult(samples[match_idx[i << 1]].name,
                                         signatures[match_idx[(i << 1) + 1]].name,
                                         match_scores[i]));
    }
    
    hipHostFree(match_scores);
    hipHostFree(match_idx);
}
