#include "hip/hip_runtime.h"
#include "kseq/kseq.h"
#include "common.h"

#define SAMPLE_MAX_LEN 200000
#define SIGNATURE_MAX_LEN 10000

__global__ void getMatches(const char* __restrict d_sample_seq, const char* __restrict d_sample_qual, const char* __restrict d_signature_seq,
        double* __restrict match_scores, unsigned int* __restrict match_idx, int* __restrict match_count) {

    const char *sample = &d_sample_seq[SAMPLE_MAX_LEN * blockIdx.x];
    const char *signature = &d_signature_seq[SIGNATURE_MAX_LEN * threadIdx.x];

    for (int i = 0; i < SAMPLE_MAX_LEN; ++i) {
        if (!sample[i]) { // Sample ended; no match
            return;
        }

        for (int j = 0; j < SIGNATURE_MAX_LEN + 1; ++j) { // +1 iteration for when signature len == SIGNATURE_MAX_LEN
            if (j == SIGNATURE_MAX_LEN || !signature[j]) { // Signature ended; found match
                const char *qual = &d_sample_qual[SAMPLE_MAX_LEN * blockIdx.x + i];
                int tot = 0;
                for (int s = 0; s < j; ++s) {
                    tot += qual[s] - 33;
                }

                int idx = atomicAdd(match_count, 1); // Each idx will be unique (no need synchronisation)
                match_idx[idx] = blockIdx.x * blockDim.x + threadIdx.x;
                match_scores[blockIdx.x * blockDim.x + threadIdx.x] = (double)tot / j;
                return;

            } else if (!sample[i + j] || sample[i + j] != 'N' && signature[j] != 'N' &&
                    sample[i + j] != signature[j]) { // No match
                break;
            }
        }
    }
}

void runMatcher(const std::vector<klibpp::KSeq>& samples, const std::vector<klibpp::KSeq>& signatures, std::vector<MatchResult>& matches) {
    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    int sample_arr_len = samples.size() * SAMPLE_MAX_LEN;
    int signature_arr_len = signatures.size() * SIGNATURE_MAX_LEN;

    // Device variables
    char *d_sample_seq, *d_sample_qual, *d_signature_seq;
    int *match_count; // 1 int storing the number of matches
    hipMallocAsync(&d_sample_seq, sample_arr_len, stream1);
    hipMemsetAsync(d_sample_seq, 0, sample_arr_len, stream1); // Initialise arrays to '\0'

    hipMallocAsync(&d_sample_qual, sample_arr_len, stream2);
    hipMemsetAsync(d_sample_qual, 0, sample_arr_len, stream2);

    hipMallocAsync(&d_signature_seq, signature_arr_len, stream3);
    hipMemsetAsync(d_signature_seq, 0, signature_arr_len, stream3);

    hipMallocAsync(&match_count, sizeof(int), stream3);
    hipMemsetAsync(match_count, 0, 1, stream3); // Initialise match_count to 0

    // Pinned memory
    double *match_scores;      // [Score_1, Score2, ...]
    unsigned int *match_idx;

    for (int i = 0; i < signatures.size(); ++i) {
        hipMemcpyAsync(&d_signature_seq[i * SIGNATURE_MAX_LEN], signatures[i].seq.c_str(), signatures[i].seq.size(), hipMemcpyHostToDevice, stream3);
    }
    for (int i = 0; i < samples.size(); ++i) {
        hipMemcpyAsync(&d_sample_seq[i * SAMPLE_MAX_LEN], samples[i].seq.c_str(), samples[i].seq.size(), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(&d_sample_qual[i * SAMPLE_MAX_LEN], samples[i].qual.c_str(), samples[i].qual.size(), hipMemcpyHostToDevice, stream2);
    }
    

    // Use hipHostMalloc() to write straight to host, since only a small number of matches should be found
    hipHostMalloc(&match_scores, sizeof(double) * samples.size() * signatures.size()); // Max possible number of matches
    hipHostMalloc(&match_idx, sizeof(unsigned int) * samples.size() * signatures.size());

    // 1 Sample per block; Each thread in block corresponds to 1 signature
    getMatches<<<samples.size(), signatures.size()>>>(d_sample_seq, d_sample_qual, d_signature_seq, match_scores, match_idx, match_count);

    int h_match_count; // Number of matches after getMatches() is done
    hipMemcpy(&h_match_count, match_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFreeAsync(match_count, stream3);
    hipFreeAsync(d_sample_seq, stream1);
    hipFreeAsync(d_sample_qual, stream2);
    hipFreeAsync(d_signature_seq, stream3);

    for (int i = 0; i < h_match_count; ++i) {
        int idx = match_idx[i];
        matches.emplace_back(MatchResult(samples[idx / signatures.size()].name,
                                         signatures[idx % signatures.size()].name,
                                         match_scores[idx]));
    }
    
    hipHostFree(match_scores);
    hipHostFree(match_idx);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
}
